#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <iostream>
#include <fstream>

#include "timer.h"

using namespace std;
typedef double (*func_t)(double x);

/* Utility function, use to do error checking.

   Use this function like this:
   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:
   checkCudaCall(hipGetLastError());
 */
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}

/*
 * Simple gauss with mu=0, sigma^1=1
 */
double gauss(double x)
{
    return exp((-1 * x * x) / 2);
}

/* Fill an double array with the function given */
void fill(float *array, int offset, int range, double sample_start,
        double sample_end, func_t f)
{
    int i;
    float dx;

    dx = (sample_end - sample_start) / range;
    for (i = 0; i < range; i++) {
        array[i + offset] = f(sample_start + i * dx);
    }
}

__global__ void waveKernel(unsigned int i_max, float* old_d, float* cur_d, float* next_d) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if ((i > 0) && (i < i_max - 1)) {
        next_d[i] = 2.0f * cur_d[i] - old_d[i] +
            0.15 * (cur_d[i-1] - (2 * cur_d[i] - cur_d[i+1]));
    } else {
        next_d[i] = 0;
    }
}


float* calcWaveCuda(int i_max, int t_max, float* old_h, float* cur_h, float* next_h) {
    int threadBlockSize = 1024;
    float *old_d, *cur_d, *next_d, *tmp;

    // allocate the vectors on the GPU
    checkCudaCall(hipMalloc((void **) &old_d, i_max * sizeof(float)));
    checkCudaCall(hipMalloc((void **) &cur_d, i_max * sizeof(float)));
    checkCudaCall(hipMalloc((void **) &next_d, i_max * sizeof(float)));

    // copy the original vectors to the GPU
    checkCudaCall(hipMemcpy(old_d, old_h, i_max*sizeof(float), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(cur_d, cur_h, i_max*sizeof(float), hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);


    // execute kernel
    hipEventRecord(start, 0);
    for (int i = 0; i < t_max; i++) {

        waveKernel<<<i_max/threadBlockSize, threadBlockSize>>>(i_max, old_d, cur_d, next_d);

        // check whether the kernel invocation was successful
        checkCudaCall(hipGetLastError());

        tmp = old_d;
        old_d = cur_d;
        cur_d = next_d;
        next_d = tmp;

        tmp = old_h;
        old_h = cur_h;
        cur_h = next_h;
        next_h = tmp;
    }
    hipEventRecord(stop, 0);

    // copy result back
    checkCudaCall(hipMemcpy(cur_h, cur_d, i_max * sizeof(float), hipMemcpyDeviceToHost));

    checkCudaCall(hipFree(old_d));
    checkCudaCall(hipFree(cur_d));
    checkCudaCall(hipFree(next_d));

    // print the time the kernel invocation took, without the copies!
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "kernel invocation took " << elapsedTime << " milliseconds" << endl;

    return cur_h;
}

int main(int argc, char *argv[])
{
    int t_max, i_max;
    float *old, *cur, *next, *ret;

    /* Parse commandline args: i_max t_max num_threads */
    if (argc < 3) {
        printf("Usage: %s i_max t_max\n", argv[0]);
        printf(" - i_max: number of discrete amplitude points, should be >2\n");
        printf(" - t_max: number of discrete timesteps, should be >=1\n");
        return EXIT_FAILURE;
    }

    /* Only accept powers of two */
    i_max = atoi(argv[1]);
    if (!(!(i_max == 0) && !(i_max & (i_max - 1)))) {
        cout << "Not a power of two" << endl;
        exit(0);
    }

    t_max = atoi(argv[2]);

    if (i_max < 3) {
        printf("argument error: i_max should be >2.\n");
        return EXIT_FAILURE;
    }
    if (t_max < 1) {
        printf("argument error: t_max should be >=1.\n");
        return EXIT_FAILURE;
    }

    /* Allocate and initialize buffers. */
   old = new float[i_max]();
   cur = new float[i_max]();
   next = new float[i_max]();

    /* Fill the first two generations */
    fill(old, 1, i_max/4, 0, 2*3.14, sin);
    fill(cur, 2, i_max/4, 0, 2*3.14, sin);

    /* Start measuring the time */
    timer vectorAddTimer("vector add timer");
    vectorAddTimer.start();

    /* Call the actual simulation and measure the time */
    ret = calcWaveCuda(i_max, t_max, old, cur, next);

    /* Print the time it took */
    vectorAddTimer.stop();
    cout << vectorAddTimer << endl;

    /* Write the output */
    ofstream returnfile;
    returnfile.open("results.txt", ios::in);
    for (int i = 0; i < i_max; i++) {
        returnfile << ret[i] << endl;
    }
    returnfile.close();

    delete[] cur;
    delete[] old;
    delete[] next;

    return EXIT_SUCCESS;
}
