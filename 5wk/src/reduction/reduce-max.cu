#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <float.h>
#include <cstdlib>
#include <iostream>

#include "timer.h"

using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:
   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:
   checkCudaCall(hipGetLastError());
 */
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}

float max_array(float a[], int num_elements)
{
    float max=FLT_MIN;
    for (int i = 0; i < num_elements; i++) {
        if (a[i] > max) {
            max = a[i];
        }
    }
    return max;
}

__global__ void reduce_max_kernel(float* input_d, float* partial_result_d, int i_max)
{
    extern __shared__ float shared_input_d[];

    unsigned global_tid = blockIdx.x * blockDim.x + threadIdx.x;

    /* load input into __shared__ memory */
    if (global_tid < i_max) {
        shared_input_d[threadIdx.x] = input_d[global_tid];
    } else {
        shared_input_d[threadIdx.x] = 0;
    }
    __syncthreads();

    /* Let every thread compare a number of the first half of the block to
       a number in the second half of the block. */
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {

        /* If the offset is smaller as the threadId, then the thread is not
           needed during computation */
        if (threadIdx.x < offset) {

            /* Get the maximum value of both cells */
            shared_input_d[global_tid] =
                ((shared_input_d[global_tid] < shared_input_d[global_tid + offset]) ?
                 shared_input_d[global_tid + offset] : shared_input_d[global_tid]);
        }

        /* Wait for all threads to update their data, so the next iteration
           can be started */
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        partial_result_d[blockIdx.x] = shared_input_d[global_tid];
    }
}


void reduce_max_cuda(int i_max, float *list_h, const int block_size, float *result_h)
{
    float *list_d = NULL;
    float *partial_result_d = NULL;

    /* number of blocks is equal to:
     * Integer division + extra block for remainder */
    const int max_blocks = (i_max/block_size) + ((i_max % block_size) ? 1 : 0);

    /* Copy the original vectors to the GPU */
    checkCudaCall(hipMalloc((void **) &list_d, i_max * sizeof(float)));
    checkCudaCall(hipMemcpy(list_d, list_h, i_max * sizeof(float), hipMemcpyHostToDevice));
    checkCudaCall(hipMalloc((void**)&partial_result_d, (max_blocks + 1) * sizeof(float)));

    /* Setup timing  */
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    /* Measure the start time  */
    hipEventRecord(start, 0);

    /* launch one kernel to compute, per-block, a partial maximum */
    reduce_max_kernel<<<max_blocks, block_size, sizeof(float) * block_size>>>
        (list_d, partial_result_d, i_max);

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    /* launch a single block to compute the maximum of the partial maximums */
    reduce_max_kernel<<<1, max_blocks, max_blocks * sizeof(float)>>>
        (partial_result_d, partial_result_d, i_max);

    /* check whether the kernel invocation was successful */
    checkCudaCall(hipGetLastError());

    /* Stop the timing */
    hipEventRecord(stop, 0);

    // copy result back
    checkCudaCall(hipMemcpy(result_h, partial_result_d, sizeof(float), hipMemcpyDeviceToHost));

    checkCudaCall(hipFree(list_d));
    checkCudaCall(hipFree(partial_result_d));

    // print the time the kernel invocation took, without the copies!
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << elapsedTime << endl;
}


int main(int argc, char* argv[])
{
    int i_max = 0;
    int block_size;
    float *result = new float[1]();
    timer maxTimer("Max Timer:");

    srand(time(NULL));

    if (argc < 3) {
        printf("Usage: %s i_max b_s\n", argv[0]);
        printf(" - i_max: number of discrete points in the list, should be >2\n");
        printf(" - b_s: block size, power of 2\n");
        return EXIT_FAILURE;
    }

    /* Only accept powers of two */
    i_max = atoi(argv[1]);
    if (!(!(i_max == 0) && !(i_max & (i_max - 1)))) {
        cout << "Not a power of two" << endl;
        exit(0);
    }
    if (i_max < 2) {
        printf("argument error: i_max should be >2.\n");
        return EXIT_FAILURE;
    }

    block_size = atoi(argv[2]);


    // make a list of floats
    float list[i_max];
    for (int i = 0; i < i_max; i++){
        list[i] = (float)rand()/((float)RAND_MAX/FLT_MAX);
    }

    maxTimer.start();
    reduce_max_cuda(i_max, list, block_size, result);
    *result = max_array(list, i_max);
    maxTimer.stop();
    /* cout << "max seq:" << max_array(list, i_max) << endl; */
    /* cout << "max cuda:" << *result << endl; */
    cout << maxTimer << endl;

    return EXIT_SUCCESS;
}
