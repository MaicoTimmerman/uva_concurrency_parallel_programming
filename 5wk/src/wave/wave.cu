#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include "timer.h"
#include <iostream>
#include <iostream>
#include <fstream>

using namespace std;
typedef double (*func_t)(double x);

/* Utility function, use to do error checking.

   Use this function like this:
   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:
   checkCudaCall(hipGetLastError());
 */
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}

/*
 * Simple gauss with mu=0, sigma^1=1
 */
double gauss(double x)
{
    return exp((-1 * x * x) / 2);
}

/* Fill an double array with the function given */
void fill(double *array, int offset, int range, double sample_start,
        double sample_end, func_t f)
{
    int i;
    float dx;

    dx = (sample_end - sample_start) / range;
    for (i = 0; i < range; i++) {
        array[i + offset] = f(sample_start + i * dx);
    }
}

__global__ void vectorAddKernel(unsigned int i_max, double* old_d, double* cur_d, double* next_d) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if ((i > 0) || (i < i_max)) {
        next_d[i] = (2*cur_d[i]) - old_d[i] + (0.15*(cur_d[i-1] - (2*cur_d[i] - cur_d[i+1])));
    } else {
        next_d[i] = 0;
    }
}


void vectorAddCuda(int i_max, int t_max, double* old_h, double* cur_h, double* next_h) {
    int threadBlockSize = 1024;
    double *tmp;

    // allocate the vectors on the GPU
    double* old_d = NULL;
    checkCudaCall(hipMalloc((void **) &old_d, i_max * sizeof(double)));
    if (old_d == NULL) {
        cout << "could not allocate memory!" << endl;
        return;
    }

    double* cur_d = NULL;
    checkCudaCall(hipMalloc((void **) &cur_d, i_max * sizeof(double)));
    if (cur_d == NULL) {
        checkCudaCall(hipFree(old_d));
        cout << "could not allocate memory!" << endl;
        return;
    }

    double* next_d = NULL;
    checkCudaCall(hipMalloc((void **) &next_d, i_max * sizeof(double)));
    if (next_d == NULL) {
        checkCudaCall(hipFree(old_d));
        checkCudaCall(hipFree(cur_d));
        cout << "could not allocate memory!" << endl;
        return;
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // copy the original vectors to the GPU
    checkCudaCall(hipMemcpy(old_d, old_h, i_max*sizeof(double), hipMemcpyHostToDevice));
    checkCudaCall(hipMemcpy(cur_d, cur_h, i_max*sizeof(double), hipMemcpyHostToDevice));

    // execute kernel
    hipEventRecord(start, 0);
    for (int i = 0; i < t_max; i++) {
        vectorAddKernel<<<i_max/threadBlockSize, threadBlockSize>>>(i_max, old_d, cur_d, next_d);
        tmp = old_d;
        old_d = cur_d;
        cur_d = next_d;
        next_d = tmp;
    }
    hipEventRecord(stop, 0);

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    checkCudaCall(hipMemcpy(cur_h, cur_d, i_max * sizeof(double), hipMemcpyDeviceToHost));

    checkCudaCall(hipFree(old_d));
    checkCudaCall(hipFree(cur_d));
    checkCudaCall(hipFree(next_d));

    // print the time the kernel invocation took, without the copies!
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    cout << "kernel invocation took " << elapsedTime << " milliseconds" << endl;
}

int main(int argc, char *argv[])
{
    int t_max, i_max;

    /* Parse commandline args: i_max t_max num_threads */
    if (argc < 3) {
        printf("Usage: %s i_max t_max\n", argv[0]);
        printf(" - i_max: number of discrete amplitude points, should be >2\n");
        printf(" - t_max: number of discrete timesteps, should be >=1\n");
        return EXIT_FAILURE;
    }

    /* Only accept powers of two */
    i_max = atoi(argv[1]);
    if (!(!(i_max == 0) && !(i_max & (i_max - 1)))) {
        cout << "Not a power of two" << endl;
        exit(0);
    }

    t_max = atoi(argv[2]);

    if (i_max < 3) {
        printf("argument error: i_max should be >2.\n");
        return EXIT_FAILURE;
    }
    if (t_max < 1) {
        printf("argument error: t_max should be >=1.\n");
        return EXIT_FAILURE;
    }

    /* Allocate and initialize buffers. */
    double *old = new double[i_max];
    double *cur = new double[i_max];
    double *next = new double[i_max];

    memset(old, 0, i_max * sizeof(double));
    memset(cur, 0, i_max * sizeof(double));
    memset(next, 0, i_max * sizeof(double));

    /* Fill the first two generations */
    fill(old, 1, i_max/4, 0, 2*3.14, sin);
    fill(cur, 2, i_max/4, 0, 2*3.14, sin);

    /* Start measuring the time */
    timer vectorAddTimer("vector add timer");
    vectorAddTimer.start();

    /* Call the actual simulation and measure the time */
    /* ret = simulate(i_max, t_max, num_threads, old, current, next); */

    /* Print the time it took */
    vectorAddTimer.stop();
    cout << vectorAddTimer << endl;

    /* Write the output */
    ofstream returnfile;
    returnfile.open("results.txt", ios::in);
    for (int i = 0; i < i_max; i++) {
        returnfile << cur[i] << endl;
    }
    returnfile.close();

    delete[] cur;
    delete[] old;
    delete[] next;

    return EXIT_SUCCESS;
}
