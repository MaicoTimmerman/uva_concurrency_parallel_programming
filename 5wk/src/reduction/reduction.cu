#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <float.h>
#include <cstdlib>
#include <iostream>

#include "timer.h"

using namespace std;

/* Utility function, use to do error checking.

   Use this function like this:
   checkCudaCall(hipMalloc((void **) &deviceRGB, imgS * sizeof(color_t)));

   And to check the result of a kernel invocation:
   checkCudaCall(hipGetLastError());
 */
static void checkCudaCall(hipError_t result) {
    if (result != hipSuccess) {
        cerr << "cuda error: " << hipGetErrorString(result) << endl;
        exit(1);
    }
}

float max_array(float a[], int num_elements)
{
    float max=FLT_MIN;
    for (int i = 0; i < num_elements; i++) {
        if (a[i] > max) {
            max = a[i];
        }
    }
    return max;
}

__global__ void vectorAddKernel(float* deviceA, float* deviceB, float* deviceResult) {
    unsigned index = blockIdx.x * blockDim.x + threadIdx.x;
    deviceResult[index] = deviceA[index] + deviceB[index];
}


float vectorAddCuda(int i_max, float* list_h) {
    /* int threadBlockSize = 512; */
    float *list_d;

    checkCudaCall(hipMalloc((void **) &list_d, i_max * sizeof(float)));

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // copy the original vectors to the GPU
    checkCudaCall(hipMemcpy(list_d, list_h, i_max * sizeof(float), hipMemcpyHostToDevice));

    // execute kernel
    hipEventRecord(start, 0);
    /* vectorAddKernel<<<n/threadBlockSize, threadBlockSize>>>(deviceA, deviceB, deviceResult); */
    hipEventRecord(stop, 0);

    // check whether the kernel invocation was successful
    checkCudaCall(hipGetLastError());

    // copy result back
    checkCudaCall(hipMemcpy(list_h, list_d, sizeof(float), hipMemcpyDeviceToHost));

    checkCudaCall(hipFree(list_d));

    // print the time the kernel invocation took, without the copies!
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);

    cout << "kernel invocation took " << elapsedTime << " milliseconds" << endl;
    return *list_h;
}


int main(int argc, char* argv[])
{
    int i_max;

    if (argc < 3) {
        printf("Usage: %s i_max\n", argv[0]);
        printf(" - i_max: number of discrete points in the list, should be >2\n");
        return EXIT_FAILURE;
    }

    /* Only accept powers of two */
    i_max = atoi(argv[1]);
    if (!(!(i_max == 0) && !(i_max & (i_max - 1)))) {
        cout << "Not a power of two" << endl;
        exit(0);
    }
    if (i_max < 2) {
        printf("argument error: i_max should be >2.\n");
        return EXIT_FAILURE;
    }


    // make a list of floats
    float list[i_max];
    for (int i = 0; i < i_max; i++){
        list[i] = (float)rand()/((float)RAND_MAX/FLT_MAX);
        printf("List[%d]: %f\n", i, list[i]);
    }

    cout << "max: " << max_array(list, i_max) << endl;

    return EXIT_SUCCESS;
}
